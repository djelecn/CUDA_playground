#include <stdio.h>
#include <hip/hip_runtime.h>
#include "gmem/1_naive.cuh"
#include "gmem/2_global_memory_coalescing.cuh"


#define M 256  // Number of rows in A and C
#define K 512   // Number of columns in A and rows in B
#define N 256  // Number of columns in B and C
#define BLOCK_SIZE 32

// Example 3x2 @ 2x4 = 3x4 -> (M x K) @ (K x N) = (M x N)
// A = [[1, 2], 
//      [3, 4], 
//      [5, 6]]

// B = [[7, 8, 9, 10],
//      [11, 12, 13, 14]]

// C = A * B = [[1*7 + 2*11, 1*8 + 2*12, 1*9 + 2*13, 1*10 + 2*14],
//              [3*7 + 4*11, 3*8 + 4*12, 3*9 + 4*13, 3*10 + 4*14],
//              [5*7 + 6*11, 5*8 + 6*12, 5*9 + 6*13, 5*10 + 6*14]]

// C = [[29, 32, 35, 38],
//      [65, 72, 79, 86],
//      [101, 112, 123, 134]]



// CPU matrix multiplication
void matmul_cpu(float *A, float *B, float *C, int m, int k, int n) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            float sum = 0.0f;
            for (int l = 0; l < k; l++) {
                sum += A[i * k + l] * B[l * n + j];
            }
            C[i * n + j] = sum;
        }
    }
}

// Initialize matrix with random values
void init_matrix(float *mat, int rows, int cols) {
    for (int i = 0; i < rows * cols; i++) {
        mat[i] = (float)rand() / RAND_MAX;
    }
}

double get_time() {
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return ts.tv_sec + ts.tv_nsec * 1e-9;
}



int main() {

    float *h_A, *h_B, *h_C;
    float *d_A, *d_B, *d_C;

    int size_A = M * K * sizeof(float);
    int size_B = K * N * sizeof(float);
    int size_C = M * N * sizeof(float);

    h_A = (float*)malloc(size_A);
    h_B = (float*)malloc(size_B);
    h_C = (float*)malloc(size_C);

    srand(time(NULL));
    init_matrix(h_A, M, K);
    init_matrix(h_B, K, N);

    float alpha = 0.5;
    float beta = 0.3;


    hipMalloc(&d_A, size_A);
    hipMalloc(&d_B, size_B);
    hipMalloc(&d_C, size_C);

    hipMemcpy(h_A, d_A, size_A, hipMemcpyHostToDevice);
    hipMemcpy(h_B, d_B, size_B, hipMemcpyHostToDevice);

    dim3 blockDim(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridDim((N + BLOCK_SIZE - 1) / BLOCK_SIZE, (M + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Warm-up runs
    printf("Performing warm-up runs...\n");
    for (int i = 0; i < 3; i++) {
        matmul_cpu(h_A, h_B, h_C, M, K, N);
        myKernels::sgemm_naive<<<gridDim, blockDim>>>(M, K, N, alpha, d_A, d_B, beta, d_C);
        hipDeviceSynchronize();
    }

    // Benchmark CPU implementation
    printf("Benchmarking CPU implementation...\n");
    double cpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        matmul_cpu(h_A, h_B, h_C, M, K, N);
        double end_time = get_time();
        cpu_total_time += end_time - start_time;
    }
    double cpu_avg_time = cpu_total_time / 20.0;


    // Benchmark GPU implementation
    printf("Benchmarking GPU implementation...\n");
    double gpu_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        myKernels::sgemm_naive<<<gridDim, blockDim>>>(M, K, N, alpha, d_A, d_B, beta, d_C);
        hipDeviceSynchronize();
        double end_time = get_time();
        gpu_total_time += end_time - start_time;
    }
    double gpu_avg_time = gpu_total_time / 20.0;

    // Benchmark GM Coalescing implementation
    printf("Benchmarking GM Coalescing implementation...\n");
    double gmc_total_time = 0.0;
    for (int i = 0; i < 20; i++) {
        double start_time = get_time();
        myKernels::gm_coalescing_sgmem<<<gridDim, blockDim>>>(M, K, N, alpha, d_A, d_B, beta, d_C);
        hipDeviceSynchronize();
        double end_time = get_time();
        gmc_total_time += end_time - start_time;
    }
    double gmc_avg_time = gmc_total_time / 20.0;

    // Print results
    printf("CPU average time: %f microseconds\n", (cpu_avg_time * 1e6f));
    printf("GPU average time: %f microseconds\n", (gpu_avg_time * 1e6f));
    printf("GM Coalescing average time: %f microseconds\n", (gmc_avg_time * 1e6f));

    printf("Speedup: %fx\n", cpu_avg_time / gmc_avg_time);

    // Free memory
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);



    return 0;
}